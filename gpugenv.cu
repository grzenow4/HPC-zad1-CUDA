#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <sstream>
#include <string>

#include "hip/hip_runtime.h"
#include "common/errors.h"
#include "common/hashmap.hh"

#define HASH_MAP_SIZE 1048576

static size_t takeIdxFromEntry(std::string& entry) {
    std::string tmp;
    std::getline(std::stringstream(entry), tmp, '\t');
    entry.erase(0, tmp.length() + 1);
    return std::stoi(tmp);
}

static Variant takeVariantFromEntry(const std::string& entry) {
    Variant variant;

    std::stringstream ss(entry);
    std::string token;

    ss >> token;
    variant.chr = std::stoi(token);

    ss >> token;
    variant.pos = std::stoi(token);

    ss >> token;
    variant.ref = token[0];

    ss >> token;
    variant.alt = token[0];

    return variant;
}

static Variant takeVariantFromVCF(const std::string& entry) {
    Variant variant;

    std::stringstream ss(entry);
    std::string token;

    ss >> token;
    variant.chr = std::stoi(token);

    ss >> token;
    variant.pos = std::stoi(token);

    ss >> token >> token;
    variant.ref = token[0];

    ss >> token;
    variant.alt = token[0];

    return variant;
}

void indexDatabase(const std::string& dbNSFP_file, const std::string& index_file) {
    std::ifstream input_file(dbNSFP_file);
    std::ofstream output_file(index_file);

    if (!input_file.is_open() || !output_file.is_open()) {
        std::cout << "Error opening a file";
        exit(EXIT_FAILURE);
    }

    std::string line;
    std::getline(input_file, line);

    int chunk_count = 0;
    HashMap *map = new HashMap(HASH_MAP_SIZE * 2);

    while (std::getline(input_file, line)) {
        chunk_count++;

        Variant variant = takeVariantFromEntry(line);
        map->insert(variant, line);

        if (chunk_count == HASH_MAP_SIZE) {
            chunk_count = 0;
            map->writeToFile(index_file);
            delete map;
            map = new HashMap(HASH_MAP_SIZE * 2);
        }
    }

    if (chunk_count > 0) {
        map->writeToFile(index_file);
        delete map;
    }

    input_file.close();
    output_file.close();
}

void matchDatabase(const std::string& file1, const std::string& file2, const std::string& file3) {
    std::ifstream input_file(file1);
    std::ifstream index_file(file2);
    std::ofstream output_file(file3);

    if (!input_file.is_open() || !index_file.is_open() || !output_file.is_open()) {
        std::cout << "Error opening a file";
        exit(EXIT_FAILURE);
    }

    std::vector<HashMap*> vec;
    std::string line;

    while (std::getline(index_file, line)) {
        if (line == "HashMap:") {
            vec.push_back(new HashMap(HASH_MAP_SIZE * 2));
            continue;
        }

        size_t idx = takeIdxFromEntry(line);
        Variant variant = takeVariantFromEntry(line);
        vec.back()->insertAt(variant, line, idx);
    }

    std::getline(input_file, line);
    std::getline(input_file, line);
    while (std::getline(input_file, line)) {
        Variant variant = takeVariantFromVCF(line);
        for (auto map: vec) {
            std::string entry = map->find(variant);
            if (entry != "") {
                output_file << entry << std::endl;
                break;
            }
        }
    }

    input_file.close();
    index_file.close();
    output_file.close();
}

__global__ void kernel() {

}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        std::cout << "Usage: " << argv[0] << " path/to/variant.vcf path/to/your/output/index path/to/matched/output.tsv\n";
        std::cout << "Index the database: " << argv[0] << " -i path/to/dbNSFP.tsv path/to/your/output/index\n";
        return 1;
    }

    std::string flag = argv[1];
    if (flag == "-i") {
        indexDatabase(argv[2], argv[3]);
        return 0;
    }

    matchDatabase(argv[1], argv[2], argv[3]);

    return 0;
}
