#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <fstream>
#include <sstream>
#include <string>

#include "hip/hip_runtime.h"
#include "common/errors.h"
#include "common/hashmap.hh"

const size_t CHUNK_SIZE = 1048576;
const size_t HASH_MAP_SIZE = 2 * CHUNK_SIZE;

static size_t takeIdxFromEntry(std::string& entry) {
    std::string tmp;
    std::getline(std::stringstream(entry), tmp, '\t');
    entry.erase(0, tmp.length() + 1);
    return std::stoi(tmp);
}

static Variant takeVariantFromEntry(const std::string& entry) {
    Variant variant;

    std::stringstream ss(entry);
    std::string token;

    ss >> token;
    variant.chr = std::stoi(token);

    ss >> token;
    variant.pos = std::stoi(token);

    ss >> token;
    variant.ref = token[0];

    ss >> token;
    variant.alt = token[0];

    return variant;
}

static Variant takeVariantFromVCF(const std::string& entry) {
    Variant variant;

    std::stringstream ss(entry);
    std::string token;

    ss >> token;
    variant.chr = std::stoi(token);

    ss >> token;
    variant.pos = std::stoi(token);

    ss >> token >> token;
    variant.ref = token[0];

    ss >> token;
    variant.alt = token[0];

    return variant;
}

void indexDatabase(const std::string& dbNSFP_file, const std::string& index_file) {
    std::ifstream input_file(dbNSFP_file);
    std::ofstream output_file(index_file);

    if (!input_file.is_open() || !output_file.is_open()) {
        std::cout << "Error opening a file";
        exit(EXIT_FAILURE);
    }

    std::string line;
    std::getline(input_file, line);

    int chunk_count = 0;
    HashMap *map = new HashMap(HASH_MAP_SIZE);

    while (std::getline(input_file, line)) {
        chunk_count++;

        Variant variant = takeVariantFromEntry(line);
        map->insert(variant, line);

        if (chunk_count == CHUNK_SIZE) {
            chunk_count = 0;
            map->writeToFile(index_file);
            delete map;
            map = new HashMap(HASH_MAP_SIZE);
        }
    }

    if (chunk_count > 0) {
        map->writeToFile(index_file);
        delete map;
    }

    input_file.close();
    output_file.close();
}

int countInputSize(const std::string& input) {
    std::ifstream input_file(input);

    if (!input_file.is_open()) {
        std::cout << "Error opening a file";
        exit(EXIT_FAILURE);
    }

    int line_count = 0;
    std::string line;

    while (std::getline(input_file, line)) {
        line_count++;
    }

    input_file.close();
    return line_count - 2;
}

void sendInputToDevice(const std::string& input, uint64_t *devInput, int size) {
    std::ifstream input_file(input);

    if (!input_file.is_open()) {
        std::cout << "Error opening a file";
        exit(EXIT_FAILURE);
    }

    uint64_t *hashes = new uint64_t[size];
    std::string line;
    std::getline(input_file, line);
    std::getline(input_file, line);
    for (int i = 0; std::getline(input_file, line); i++) {
        assert(i < size);
        hashes[i] = hashVariant(takeVariantFromVCF(line));
    }

    HANDLE_ERROR(hipMemcpy(devInput, hashes, size * sizeof(uint64_t), hipMemcpyHostToDevice));

    delete[] hashes;
    input_file.close();
}

__global__ void kernel(uint64_t *out, uint64_t *input, int input_size, uint64_t *map) {
    int idx = threadIdx.x;

    for (int i = 0; i < input_size; i++) {
        out[i] = 0;

        uint64_t hash = input[i];
        size_t index = static_cast<size_t>(hash & (HASH_MAP_SIZE - 1));

        while (map[index] != 0) {
            if (hash == map[index]) {
                out[i] = index;
                break;
            }
            index++;
            if (index == HASH_MAP_SIZE) {
                index = 0;
            }
        }
    }
}

void invokeKernel(uint64_t *devOut, uint64_t *devInput, int input_size, uint64_t *devMap, float &elapsedTime) {
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipEventRecord(start, 0));
    kernel<<<1, input_size>>>(devOut, devInput, input_size, devMap);
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float time;
    HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
    elapsedTime += time;

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
}

void matchDatabase(const std::string& input,
                   const std::string& index,
                   const std::string& output) {
    std::ifstream index_file(index);
    std::ofstream output_file(output);

    if (!index_file.is_open() || !output_file.is_open()) {
        std::cout << "Error opening a file";
        exit(EXIT_FAILURE);
    }

    int input_size = countInputSize(input);
    float elapsedTime = 0;

    uint64_t *out = (uint64_t*) malloc(input_size * sizeof(uint64_t));
    uint64_t *devInput;
    uint64_t *devOut;
    uint64_t *devMap;

    HANDLE_ERROR(hipMalloc((void**)&devInput, input_size * sizeof(uint64_t)));
    HANDLE_ERROR(hipMalloc((void**)&devOut, input_size * sizeof(uint64_t)));
    HANDLE_ERROR(hipMalloc((void**)&devMap, HASH_MAP_SIZE * sizeof(uint64_t)));

    sendInputToDevice(input, devInput, input_size);

    HashMap *map = nullptr;
    std::string line;

    while (std::getline(index_file, line)) {
        if (line == "HashMap:") {
            if (map != nullptr) {
                uint64_t *tmp = map->dumpVariants();
                HANDLE_ERROR(hipMemcpy(devMap, tmp, HASH_MAP_SIZE * sizeof(uint64_t), hipMemcpyHostToDevice));

                invokeKernel(devOut, devInput, input_size, devMap, elapsedTime);

                HANDLE_ERROR(hipMemcpy(out, devOut, input_size * sizeof(uint64_t), hipMemcpyDeviceToHost));
                for (int i = 0; i < input_size; i++) {
                    if (out[i] != 0) {
                        output_file << map->entries[out[i]].dbEntry << "\n";
                    }
                }

                free(tmp);
                free(map);
            }
            map = new HashMap(HASH_MAP_SIZE);
        } else {
            size_t idx = takeIdxFromEntry(line);
            Variant variant = takeVariantFromEntry(line);
            map->insertAt(variant, line, idx);
        }
    }
    uint64_t *tmp = map->dumpVariants();
    HANDLE_ERROR(hipMemcpy(devMap, tmp, HASH_MAP_SIZE * sizeof(uint64_t), hipMemcpyHostToDevice));

    invokeKernel(devOut, devInput, input_size, devMap, elapsedTime);

    HANDLE_ERROR(hipMemcpy(out, devOut, input_size * sizeof(uint64_t), hipMemcpyDeviceToHost));
    for (int i = 0; i < input_size; i++) {
        if (out[i] != 0) {
            output_file << map->entries[out[i]].dbEntry << "\n";
        }
    }

    free(tmp);
    free(map);
    free(out);

    std::cout << "Total GPU execution time: " << elapsedTime << " ms\n";

    HANDLE_ERROR(hipFree(devInput));
    HANDLE_ERROR(hipFree(devOut));
    HANDLE_ERROR(hipFree(devMap));

    index_file.close();
    output_file.close();
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        std::cout << "Usage: " << argv[0] << " path/to/variant.vcf path/to/your/output/index path/to/matched/output.tsv\n";
        std::cout << "Index the database: " << argv[0] << " -i path/to/dbNSFP.tsv path/to/your/output/index\n";
        return 1;
    }

    std::string flag = argv[1];
    if (flag == "-i") {
        indexDatabase(argv[2], argv[3]);
        return 0;
    }

    matchDatabase(argv[1], argv[2], argv[3]);

    return 0;
}
