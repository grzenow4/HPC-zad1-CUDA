#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime.h"
#include "common/errors.h"
#include "common/hashmap.h"

void indexDatabase(char *dbNSFP_file, char *index_file) {
    FILE *input_file = fopen(dbNSFP_file, "r");
    FILE *output_file = fopen(index_file, "wb");

    if (!input_file || !output_file) {
        printf("Error opening a file");
        exit(EXIT_FAILURE);
    }

    HashMap *map = createHashMap();

    char *line = NULL;
    size_t len = 0;
    Variant variant;

    // Discard the header line
    getline(&line, &len, input_file);

    while (getline(&line, &len, input_file) != -1) {
        char *line_copy = strdup(line);
        if (line_copy == NULL) {
            printf("Memory allocation failed");
            exit(EXIT_FAILURE);
        }

        char *token = strtok(line, "\t");
        variant.chr = atoi(token);

        token = strtok(NULL, "\t");
        variant.pos = atoi(token);

        token = strtok(NULL, "\t");
        variant.ref = token[0];

        token = strtok(NULL, "\t");
        variant.alt = token[0];

        insertHashMap(map, variant, line_copy);
    }

    printMap(map);
    fwrite(map->entries, sizeof(HashEntry), map->size, output_file);

    free(line);
    destroyHashMap(map);
    fclose(input_file);
    fclose(output_file);
}

__global__ void kernel() {

}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s path/to/variant.vcf path/to/your/output/index path/to/matched/output.tsv\n", argv[0]);
        printf("Index the database: %s -i path/to/dbNSFP.tsv path/to/your/output/index\n", argv[0]);
        return 1;
    }

    if (strcmp(argv[1], "-i") == 0) {
        indexDatabase(argv[2], argv[3]);
        return 0;
    }

    FILE *input_file = fopen(argv[1], "r");
    FILE *index_file = fopen(argv[2], "r");
    FILE *output_file = fopen(argv[3], "wb");

    if (!input_file || !index_file || !output_file) {
        printf("Error opening a file");
        exit(EXIT_FAILURE);
    }

    fclose(input_file);
    fclose(index_file);
    fclose(output_file);

    return 0;
}
